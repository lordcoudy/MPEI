#include <iostream>
#include <vector>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>

#define N 512
#define M 60*7
#define bigN 65536          // 64 KB
#define coeff 64
#define CAP (bigN / coeff)  // 4 KB for 4 arrays

using namespace std;
using namespace std::chrono;

// Macros to check CUDA errors that are not shown by compiler (simple check of cuda success)
#define checkCUDA(expression) { gpuAssert((expression), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t status, const char* file, int line)
{
    if (status != hipSuccess)
    {
        std::cerr << "GPUassert: " << hipGetErrorString(status) << ", " << file << ", " << line << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

// Функция для расчёта времени выполнения функции
template<typename Func>
double measureExecutionTime(Func func) {
    auto startTime = high_resolution_clock::now();
    func();
    auto endTime = high_resolution_clock::now();
    auto duration = duration_cast<nanoseconds>(endTime - startTime).count();
    return duration / 1e6;                      // Перевод в миллисекунды
}

int A[N], B[N];
double X[N], C[N];

int bigA[bigN], bigB[bigN];
double bigX[bigN], bigC[bigN];

void fillArrays(int *a, int *b, double *c, double *x) {
    for (int i = 0; i < N; i++) {
        a[i] = 2;
        b[i] = 1;
        c[i] = 1.5;
        x[i] = 1.0;
    }
}

void fillBigArrays(int *a, int *b, double *c, double *x) {
    for (int i = 0; i < bigN; i++) {
        a[i] = 2;
        b[i] = 1;
        c[i] = 1.5;
        x[i] = 1.0;
    }
}

void hostWork(const int *a, const int *b, const double *c, double *x) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            x[i] = (double) a[i] * (x[i] + b[j]) / c[j];
        }
    }
}

__global__ void globalMemoryGPU(const int *a, const int *b, const double *c, double *x, int size) {
    uint8_t i = threadIdx.x + blockIdx.x * blockDim.x;
    for (int y = 0; y < size; y++) {
        int bigIndex = i * size + y;
        for (int j = 0; j < M; j++) {
            x[bigIndex] = (double) a[bigIndex] * (x[bigIndex] + b[bigIndex]) / c[bigIndex];
        }
    }
}

__global__ void sharedMemoryGPU(const int *a, const int *b, const double *c, double *x, int size) {
    __shared__ double sharedX[N];
    __shared__ int sharedA[N];
    __shared__ int sharedB[N];
    __shared__ double sharedC[N];
    uint8_t i = threadIdx.x + blockIdx.x * blockDim.x;
    for (int y = 0; y < size; y++) {
        int bigIndex = i * size + y;
        sharedA[bigIndex] = a[bigIndex];
        sharedB[bigIndex] = b[bigIndex];
        sharedC[bigIndex] = c[bigIndex];
        sharedX[bigIndex] = x[bigIndex];
        __syncthreads();
        for (int j = 0; j < M; j++) {
            sharedX[bigIndex] = (double) sharedA[bigIndex] * (sharedX[bigIndex] + sharedB[bigIndex]) / sharedC[bigIndex];
        }
        __syncthreads();
        x[bigIndex] = sharedX[bigIndex];
    }
}

__constant__ int constantA[N];
__constant__ int constantB[N];
__constant__ double constantC[N];

__global__ void constantMemoryGPU(double *x, int size) {
    __shared__ double sharedX[N];
    uint8_t i = threadIdx.x + blockIdx.x * blockDim.x;
    for (int y = 0; y < size; y++) {
        int bigIndex = i * size + y;
        sharedX[bigIndex] = x[bigIndex];
        __syncthreads();
        for (int j = 0; j < M; j++) {
            sharedX[bigIndex] = (double) constantA[bigIndex] * (sharedX[bigIndex] + constantB[bigIndex]) / constantC[bigIndex];
        }
        __syncthreads();
        x[bigIndex] = sharedX[bigIndex];
    }
}

__global__ void sharedMemoryCapGPU(const int *a, const int *b, const double *c, double *x, int size, int cap_size) {
    uint8_t i = threadIdx.x + blockIdx.x * blockDim.x;

    for (int r = 0; r < coeff; r++) {
        __shared__ double sharedX[CAP];
        __shared__ int sharedA[CAP];
        __shared__ int sharedB[CAP];
        __shared__ double sharedC[CAP];

        for (int y = 0; y < cap_size; y++) {
            int capIndex = i * cap_size + y;
            int bigIndex = capIndex + r * CAP;
            sharedA[capIndex] = a[bigIndex];
            sharedB[capIndex] = b[bigIndex];
            sharedC[capIndex] = c[bigIndex];
            sharedX[capIndex] = x[bigIndex];
            __syncthreads();
            for (int j = 0; j < M; j++) {
                sharedX[capIndex] = (double) sharedA[capIndex] * (sharedX[capIndex] + sharedB[capIndex]) / sharedC[capIndex];
            }
            __syncthreads();
            x[bigIndex] = sharedX[capIndex];
        }
    }
}

__constant__ int constantBigA[CAP];
__constant__ int constantBigB[CAP];
__constant__ double constantBigC[CAP];

__global__ void constantMemoryCapGPU(double *x, int size, int cap_size, int cap_index) {
   uint8_t i = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ double sharedX[CAP];
   for (int y = 0; y < cap_size; y++) {
        int capIndex = i * cap_size + y;
        int bigIndex = capIndex + cap_index * CAP;
        sharedX[capIndex] = x[bigIndex];
        __syncthreads();
        for (int j = 0; j < M; j++) {
            sharedX[capIndex] = (double) constantBigA[capIndex] * (sharedX[capIndex] + constantBigB[capIndex]) / constantBigC[capIndex];
        }
        __syncthreads();
        x[bigIndex] = sharedX[capIndex];
    }
}

void runCUDA(const int GRID_SIZE, const int BLOCK_SIZE) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    int *d_A, *d_B;
    double *d_C, *d_X;
    const int forSize = N / (GRID_SIZE * BLOCK_SIZE);
    const dim3 block(BLOCK_SIZE);                                   // Размер блока (одномерный)
    const dim3 grid(GRID_SIZE);                                     // Размер сетки из блоков (одномерная)

    fillArrays(A, B, C, X);

    cout << "\n-------------------------- GLOBAL PART STARTED --------------------------\n";
    checkCUDA(hipMalloc(&d_A, N * sizeof(int)));
    checkCUDA(hipMalloc(&d_B, N * sizeof(int)));
    checkCUDA(hipMalloc(&d_C, N * sizeof(double)));
    checkCUDA(hipMalloc(&d_X, N * sizeof(double)));

    checkCUDA(hipMemcpy(d_A, A, N * sizeof(int), hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_B, B, N * sizeof(int), hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_C, C, N * sizeof(double), hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_X, X, N * sizeof(double), hipMemcpyHostToDevice));

    hipEventRecord(start);                                         // Запуск таймера
    globalMemoryGPU <<< grid, block >>>(d_A, d_B, d_C, d_X, forSize);   // Запуск процедуры
    checkCUDA(hipPeekAtLastError());                               // Проверка на ошибки
    hipEventRecord(stop);                                          // Остановка таймера

    checkCUDA(hipMemcpy(X, d_X, N * sizeof(double), hipMemcpyDeviceToHost));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_X);

    hipEventSynchronize(stop);                                     // Синхронизация таймера
    float time;                                                     // Время
    hipEventElapsedTime(&time, start, stop);                       // Расчёт времени
    ofstream globalTime("../gpu_global.csv", std::ios::app);                  // Открытие файла
    cout << "\nProcessing time for global memory GPU (ms): " << time << "\n";     // Вывод в поток
    // Вывод в файл
    globalTime << N << "," << BLOCK_SIZE << "," << GRID_SIZE << "," << time << "\n";
    globalTime.close();                                                // Закрытие файла
    cout << "\n-------------------------- GLOBAL PART ENDED --------------------------\n";
    hipDeviceSynchronize();

    cout << "\n-------------------------- SHARED PART STARTED --------------------------\n";
    fillArrays(A, B, C, X);

    checkCUDA(hipMalloc(&d_A, N * sizeof(int)));
    checkCUDA(hipMalloc(&d_B, N * sizeof(int)));
    checkCUDA(hipMalloc(&d_C, N * sizeof(double)));
    checkCUDA(hipMalloc(&d_X, N * sizeof(double)));

    checkCUDA(hipMemcpy(d_A, A, N * sizeof(int), hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_B, B, N * sizeof(int), hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_C, C, N * sizeof(double), hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_X, X, N * sizeof(double), hipMemcpyHostToDevice));

    hipEventRecord(start);                                         // Запуск таймера
    sharedMemoryGPU <<< grid, block >>>(d_A, d_B, d_C, d_X, forSize);   // Запуск процедуры
    checkCUDA(hipPeekAtLastError());                               // Проверка на ошибки
    hipEventRecord(stop);                                          // Остановка таймера

    checkCUDA(hipMemcpy(X, d_X, N * sizeof(double), hipMemcpyDeviceToHost));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_X);

    hipEventSynchronize(stop);                                     // Синхронизация таймера
    hipEventElapsedTime(&time, start, stop);                       // Расчёт времени
    ofstream sharedTime("../gpu_shared.csv", std::ios::app);                  // Открытие файла
    cout << "\nProcessing time for shared memory GPU (ms): " << time << "\n";     // Вывод в поток
    // Вывод в файл
    sharedTime << N << "," << BLOCK_SIZE << "," << GRID_SIZE << "," << time << "\n";
    sharedTime.close();                                                // Закрытие файла
    cout << "\n-------------------------- SHARED PART FINISHED --------------------------\n";
    hipDeviceSynchronize();

    cout << "\n-------------------------- CONSTANT PART STARTED --------------------------\n";
    fillArrays(A, B, C, X);

    checkCUDA(hipMalloc(&d_X, N * sizeof(double)));

    checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(constantA), A, N * sizeof(int)));
    checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(constantB), B, N * sizeof(int)));
    checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(constantC), C, N * sizeof(double)));
    checkCUDA(hipMemcpy(d_X, X, N * sizeof(double), hipMemcpyHostToDevice));

    hipEventRecord(start);                                         // Запуск таймера
    constantMemoryGPU <<< grid, block >>>(d_X, forSize);   // Запуск процедуры
    checkCUDA(hipPeekAtLastError());                               // Проверка на ошибки
    hipEventRecord(stop);                                          // Остановка таймера

    checkCUDA(hipMemcpy(X, d_X, N * sizeof(double), hipMemcpyDeviceToHost));
    hipFree(d_X);

    hipEventSynchronize(stop);                                     // Синхронизация таймера
    hipEventElapsedTime(&time, start, stop);                       // Расчёт времени
    ofstream constantTime("../gpu_const.csv", std::ios::app);                  // Открытие файла
    cout << "\nProcessing time for constant memory GPU (ms): " << time << "\n";     // Вывод в поток
    // Вывод в файл
    constantTime << N << "," << BLOCK_SIZE << "," << GRID_SIZE << "," << time << "\n";
    constantTime.close();                                                // Закрытие файла
    cout << "\n-------------------------- CONSTANT PART ENDED --------------------------\n";
    hipDeviceSynchronize();

    cout << "\n-------------------------- SHARED CAP PART STARTED --------------------------\n";
    fillBigArrays(bigA, bigB, bigC, bigX);
    const int forBigSize = bigN / (GRID_SIZE * BLOCK_SIZE);
    const int forCapSize = CAP / (GRID_SIZE * BLOCK_SIZE);

    checkCUDA(hipMalloc(&d_A, bigN * sizeof(int)));
    checkCUDA(hipMalloc(&d_B, bigN * sizeof(int)));
    checkCUDA(hipMalloc(&d_C, bigN * sizeof(double)));
    checkCUDA(hipMalloc(&d_X, bigN * sizeof(double)));

    checkCUDA(hipMemcpy(d_A, bigA, bigN * sizeof(int), hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_B, bigB, bigN * sizeof(int), hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_C, bigC, bigN * sizeof(double), hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_X, bigX, bigN * sizeof(double), hipMemcpyHostToDevice));

    hipEventRecord(start);                                         // Запуск таймера
    sharedMemoryCapGPU <<< grid, block >>>(d_A, d_B, d_C, d_X, forBigSize, forCapSize);   // Запуск процедуры
    checkCUDA(hipPeekAtLastError());                               // Проверка на ошибки
    hipEventRecord(stop);                                          // Остановка таймера

    checkCUDA(hipMemcpy(bigX, d_X, bigN * sizeof(double), hipMemcpyDeviceToHost));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_X);

    hipEventSynchronize(stop);                                     // Синхронизация таймера
    hipEventElapsedTime(&time, start, stop);                       // Расчёт времени
    ofstream capSharedTime("../gpu_shared_cap.csv", std::ios::app);                  // Открытие файла
    cout << "\nProcessing time for shared cap memory GPU (ms): " << time << "\n";     // Вывод в поток
    // Вывод в файл
    capSharedTime << bigN << "," << BLOCK_SIZE << "," << GRID_SIZE << "," << time << "\n";
    capSharedTime.close();                                                // Закрытие файла
    cout << "\n-------------------------- SHARED CAP PART ENDED --------------------------\n";
    hipDeviceSynchronize();

    cout << "\n-------------------------- CONSTANT CAP PART STARTED --------------------------\n";
    fillBigArrays(bigA, bigB, bigC, bigX);

    checkCUDA(hipMalloc(&d_X, bigN * sizeof(double)));

    checkCUDA(hipMemcpy(d_X, bigX, bigN * sizeof(double), hipMemcpyHostToDevice));


    /// CONSTANT CAP
    hipEventRecord(start);                                         // Запуск таймера
    for (int i = 0; i < coeff; ++i)
    {
        checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(constantBigA), bigA + coeff, CAP * sizeof(int)));
        checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(constantBigB), bigB + coeff, CAP * sizeof(int)));
        checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(constantBigC), bigC + coeff, CAP * sizeof(double)));
        constantMemoryCapGPU <<< grid, block >>>(d_X, forBigSize, forCapSize, coeff);   // Запуск процедуры
    }
    checkCUDA(hipPeekAtLastError());                               // Проверка на ошибки
    hipEventRecord(stop);                                          // Остановка таймера

    checkCUDA(hipMemcpy(X, d_X, bigN * sizeof(double), hipMemcpyDeviceToHost));
    hipFree(d_X);

    hipEventSynchronize(stop);                                     // Синхронизация таймера
    hipEventElapsedTime(&time, start, stop);                       // Расчёт времени
    ofstream capConstantTime("../gpu_const_cap.csv", std::ios::app);                  // Открытие файла
    cout << "\nProcessing time for constant cap memory GPU (ms): " << time << "\n";     // Вывод в поток
    // Вывод в файл
    capConstantTime << bigN << "," << BLOCK_SIZE << "," << GRID_SIZE << "," << time << "\n";
    capConstantTime.close();                                                // Закрытие файла
    cout << "\n-------------------------- CONSTANT CAP PART ENDED --------------------------\n";
}

int main() {
    cout << "\n-------------------------- PROCESSING STARTED --------------------------\n";
    fillArrays(A, B, C, X);
    cout << "\n-------------------------- CPU PART STARTED --------------------------\n";
    double time = measureExecutionTime([&](){ hostWork(A, B, C, X);});
    ofstream cpuTime("../cpu.csv", std::ios::app);
    cout << "\nProcessing time for CPU (ms): " << time << "\n";
    cpuTime << N << "," << time << "\n";
    cpuTime.close();
    cout << "\n-------------------------- CPU PART FINISHED --------------------------\n";

    vector<int> block_sizes = {1, 4, 32, 128, 256};
    vector<int> grid_sizes = {1, 2, 4};

    cout << "\n-------------------------- CUDA PART STARTED --------------------------\n";
    for (const auto &grid : grid_sizes) {
        for (const auto &block : block_sizes) {
            runCUDA(grid, block);
        }
    }
    cout << "\n-------------------------- CUDA PART FINISHED --------------------------\n";
    cout << "\n-------------------------- PROCESSING FINISHED --------------------------\n";

    return 0;
}
