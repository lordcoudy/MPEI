#include "hip/hip_runtime.h"
// ВНИМАНИЕ
// ЛОВУШКА
// В КОДЕ ДОПУЩЕНЫ ОШИБКИ, ВЛИЯЮЩИЕ НА РАБОТУ АЛГОРИТМА
// КТО ПОЧИНИТ - ТОТ МОЛОДЕЦ

#include <iostream>
#include <fstream>
#include <vector>
#include <opencv2/opencv.hpp>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

using namespace std;
using namespace cv;
using namespace std::chrono;

#define PI 3.141592653
#define radSigCoeff 3
#define BLOCK_SIZE 32

// Запрограммировать алгоритм Гауссова размытия. Этот фильтр используется для подавления шума.

#define checkCUDA(expression) { gpuAssert((expression), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t status, const char* file, int line)
{
    if (status != hipSuccess)
    {
        std::cerr << "GPUassert: " << hipGetErrorString(status) << ", " << file << ", " << line << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

// Функция для расчёта времени выполнения функции
template<typename Func>
double measureExecutionTime(Func func) {
    auto startTime = high_resolution_clock::now();
    func();
    auto endTime = high_resolution_clock::now();
    auto duration = duration_cast<nanoseconds>(endTime - startTime).count();
    return duration / 1e6;                      // Перевод в миллисекунды
}

void calculateKernel(float *kernel, int sigma) {
    int radius = radSigCoeff * sigma;
    for (int i = 0; i <= radius; i++) {
        kernel[i + radius] = kernel[radius - i] = exp(
                -(pow(i, 2) / (2 * pow(sigma,2)))
                ) / (
                        sqrt(2 * PI * pow(sigma, 2))
                        );
    }
}

__global__ void gaussianBlurRowsGPU(
        const unsigned char *srcImage, unsigned char *dstImage, int width, int height,
        const float *kernel, int kernel_radius) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if ((row >= 0) && (col >= 0) && (row < height) && (col < width))
    {
        float sum = 0.0f;
        float coeffSum = 0.0f;
        float coeff = 0.0f;
        for (int i = -kernel_radius; i <= kernel_radius; i++) {
            coeff = kernel[i + kernel_radius];
            int currentCol = col + i;
            if ((row * width + currentCol) >= 0 && (row * width + currentCol) < (width * height)) {
                sum += srcImage[row * width + currentCol] * coeff;
                coeffSum += coeff;
            }
        }
        dstImage[row * width + col] = sum / (coeffSum > 0 ? coeffSum : 1);
    }
}

__global__ void gaussianBlurColsGPU(
        const unsigned char *srcImage, unsigned char *dstImage, int width, int height,
        const float *kernel, int kernel_radius) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if ((row >= 0) && (col >= 0) && (row < height) && (col < width))
    {
        float sum = 0.0f;
        float coeffSum = 0.0f;
        float coeff = 0.0f;
        for (int i = -kernel_radius; i <= kernel_radius; i++) {
            coeff = kernel[i + kernel_radius];
            int currentRow = row + i;
            if ((col * height + currentRow) >= 0 && (col * height + currentRow) < (width * height)) {
                sum += srcImage[col * height + currentRow] * coeff;
                coeffSum += coeff;
            }
        }
        dstImage[col * height + row] = sum / (coeffSum > 0 ? coeffSum : 1);
    }
}

void runCuda(Mat &srcImage, Mat &dstImage, float *kernel, int &kernel_radius) {
    // Use cuda event to catch time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /*Get image size*/
    int height = srcImage.rows;										// number of input image rows
    int width = srcImage.cols;										// number of input image columns

    const long long imageSize = height * width;
    unsigned char* d_input = nullptr, * d_output = nullptr, * d_tmp = nullptr;
    float* d_kernel = nullptr;

    Mat tmpImage = srcImage.clone();
    // Allocate device memory
    checkCUDA(hipMalloc(&d_input, imageSize * sizeof(unsigned char)));
    checkCUDA(hipMalloc(&d_tmp, imageSize * sizeof(unsigned char)));
    checkCUDA(hipMalloc(&d_output, imageSize * sizeof(unsigned char)));
    checkCUDA(hipMalloc(&d_kernel, (kernel_radius * 2 + 1) * sizeof(float)));


    // Copy data from OpenCV input image to device memory
    checkCUDA(hipMemcpy(d_input, srcImage.ptr(), imageSize * sizeof(unsigned char), hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_kernel, kernel, (kernel_radius * 2 + 1) * sizeof(float), hipMemcpyHostToDevice));

    // Specify block size
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((int)ceil((float)width / block.x), (int)ceil((float)srcImage.rows / block.y));

    // Start time
    hipEventRecord(start);

    // Launch kernel
    gaussianBlurRowsGPU<<<grid, block>>>(d_input, d_tmp, width, height, d_kernel, kernel_radius);
    gaussianBlurColsGPU<<<grid, block>>>(d_tmp, d_output, width, height, d_kernel, kernel_radius);
    checkCUDA(hipPeekAtLastError());

    // End time
    hipEventRecord(stop);

    //Copy data from device memory to output image
    checkCUDA(hipMemcpy(dstImage.ptr(), d_output, imageSize * sizeof(unsigned char), hipMemcpyDeviceToHost));

    //Free the device memoryS
    hipFree(d_input);
    hipFree(d_tmp);
    hipFree(d_output);
    hipFree(d_kernel);
    hipEventSynchronize(stop);

    float time;                                                     // Время
    hipEventElapsedTime(&time, start, stop);                       // Расчёт времени
    ofstream globalTime("../gpu.csv", std::ios::app);                  // Открытие файла
    cout << "\nProcessing time for shared memory GPU (ms): " << time << "\n";     // Вывод в поток
    // Вывод в файл
    globalTime << kernel_radius << "," << width << "," << height << "," << time << "\n";
    globalTime.close();                                                // Закрытие файла
}

void gaussianBlurRowsCPU(unsigned char *srcImage, unsigned char *dstImage, int width, int height,
                         float *kernel, int kernel_radius)
{
    for (int y = 0; y < height; y++)
        for (int x = 0; x < width; x++) {
            float sum = 0;
            for (int k = -kernel_radius; k <= kernel_radius; k++) {
                int d = x + k;
                if (d >= 0 && d < width)
                    sum += srcImage[y * width + d] * kernel[kernel_radius - k];
            }
            dstImage[y * width + x] = sum;
        }
}

void gaussianBlurColsCPU(unsigned char *srcImage, unsigned char *dstImage, int width, int height,
                         float *kernel, int kernel_radius)
{
    for (int y = 0; y < height; y++)
        for (int x = 0; x < width; x++) {
            float sum = 0;
            for (int k = -kernel_radius; k <= kernel_radius; k++) {
                int d = y + k;
                if (d >= 0 && d < height)
                    sum += srcImage[d * width + x] * kernel[kernel_radius - k];
            }
            dstImage[y * width + x] = sum;
        }
}

void runCpu(Mat &srcImage, Mat &dstImage, float *kernel, int &kernel_radius) {
    Mat tmpImage = srcImage.clone();
    gaussianBlurRowsCPU(srcImage.ptr(), tmpImage.ptr(), srcImage.cols, srcImage.rows, kernel, kernel_radius);
    gaussianBlurColsCPU(tmpImage.ptr(), dstImage.ptr(), srcImage.cols, srcImage.rows, kernel, kernel_radius);
}

int main() {
    vector<int> sigmas = {1, 2, 3, 4, 5};
    vector<int> radii = {sigmas[0]*radSigCoeff, sigmas[1]*radSigCoeff, sigmas[2]*radSigCoeff, sigmas[3]*radSigCoeff, sigmas[4]*radSigCoeff};
    vector<float*> kernels;

    for (int i = 0; i < sigmas.size(); i++) {
        auto *kernel = new float[radii[i]*2 + 1];
        calculateKernel(kernel, sigmas[i]);
        kernels.push_back(kernel);
    }

    vector<string> paths = {
            "image1.tiff",
            "image2.tiff",
            "image3.tiff",
    };
    
    for (const auto & path : paths)
    {
        Mat srcImage = imread("../"+path, IMREAD_GRAYSCALE);
        if (srcImage.empty()) {
            cerr << "Could not open or find the image" << endl;
            return -1;
        }

        Mat dstImage = srcImage.clone();
        Mat dstImageCPU = srcImage.clone();

        ofstream globalTime("../gpu.csv", std::ios::app);
        globalTime << "radius" << "," << "width" << "," << "height" << "," << "time(ms)" << "\n";
        globalTime.close();

        ofstream cpuTime("../cpu.csv", std::ios::app);
        cpuTime << "radius" << "," << "width" << "," << "height" << "," << "time(ms)" << "\n";
        cpuTime.close();

        double time = 0;
        for (int i = 0; i < kernels.size(); i++) {
            time = measureExecutionTime([&](){ runCpu(srcImage, dstImageCPU, kernels[i], radii[i]); });
            ofstream cpuTime("../cpu.csv", std::ios::app);
            cout << "\nProcessing time for CPU (ms): " << time << "\n";
            cpuTime << radii[i] << "," << srcImage.cols << "," << srcImage.rows << "," << time << "\n";
            cpuTime.close();
            string output_cpu = "../cpu_" + to_string(radii[i]) + "_" + path;
            imwrite(output_cpu, dstImageCPU);

            runCuda(srcImage, dstImage, kernels[i], radii[i]);
            string output_gpu = "../gpu_" + to_string(radii[i]) + "_" + path;
            imwrite(output_gpu, dstImage);
        }
    }


    for (auto & kernel : kernels) {
        delete[] kernel;
    }

    return 0;
}
