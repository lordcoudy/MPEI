#include "hip/hip_runtime.h"
#pragma region Libs
#include <iostream>
#include <vector>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
using namespace std;
using namespace std::chrono;
#pragma endregion

// Значение степени по умолчанию
const int m = 3;

#pragma region CUDA checking
// Макрос и процедура для проверки выполнения CUDA кода
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
#pragma endregion

#pragma region CPU time count
// Функция для расчёта времени выполнения функции
template<typename Func>
double measureExecutionTime(Func func) {
    auto startTime = high_resolution_clock::now();
    func();
    auto endTime = high_resolution_clock::now();
    auto duration = duration_cast<nanoseconds>(endTime - startTime).count();
    return duration / 1e6;                      // Перевод в миллисекунды
}
#pragma endregion

#pragma region Info
// Вся информация о CUDA и GPU
void AboutDevice()
{
    printf("\n-------------------------- BEGIN OF DEVICE INFO --------------------------\n");
    printf("Starting...\n\n");

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
               static_cast<int>(error_id), hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
    }
    else {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }

    int dev = 0, driverVersion = 0, runtimeVersion = 0;

    hipSetDevice(dev);
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, dev);

    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

    // Console log
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
           driverVersion / 1000, (driverVersion % 100) / 10,
           runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
           deviceProp.major, deviceProp.minor);

    char msg[256];

    sprintf_s(msg, sizeof(msg),
              "  Total amount of global memory:                 %.0f MBytes "
              "(%llu bytes)\n",
              static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f),
              (unsigned long long)deviceProp.totalGlobalMem);
    printf("%s", msg);

    printf("  (%03d) Multiprocessors\n",
           deviceProp.multiProcessorCount);
    printf(
            "  GPU Max Clock rate:                            %.0f MHz (%0.2f "
            "GHz)\n",
            deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
    printf("  Memory Clock rate:                             %.0f Mhz\n",
           deviceProp.memoryClockRate * 1e-3f);
    printf("  Memory Bus Width:                              %d-bit\n",
           deviceProp.memoryBusWidth);

    if (deviceProp.l2CacheSize) {
        printf("  L2 Cache Size:                                 %d bytes\n",
               deviceProp.l2CacheSize);
    }

    printf(
            "  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, "
            "%d), 3D=(%d, %d, %d)\n",
            deviceProp.maxTexture1D, deviceProp.maxTexture2D[0],
            deviceProp.maxTexture2D[1], deviceProp.maxTexture3D[0],
            deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
    printf(
            "  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
            deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
    printf(
            "  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d "
            "layers\n",
            deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
            deviceProp.maxTexture2DLayered[2]);

    printf("  Total amount of constant memory:               %zu bytes\n",
           deviceProp.totalConstMem);
    printf("  Total amount of shared memory per block:       %zu bytes\n",
           deviceProp.sharedMemPerBlock);
    printf("  Total shared memory per multiprocessor:        %zu bytes\n",
           deviceProp.sharedMemPerMultiprocessor);
    printf("  Total number of registers available per block: %d\n",
           deviceProp.regsPerBlock);
    printf("  Warp size:                                     %d\n",
           deviceProp.warpSize);
    printf("  Maximum number of threads per multiprocessor:  %d\n",
           deviceProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of threads per block:           %d\n",
           deviceProp.maxThreadsPerBlock);
    printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
           deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
           deviceProp.maxThreadsDim[2]);
    printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
           deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
           deviceProp.maxGridSize[2]);
    printf("  Maximum memory pitch:                          %zu bytes\n",
           deviceProp.memPitch);
    printf("  Texture alignment:                             %zu bytes\n",
           deviceProp.textureAlignment);
    printf(
            "  Concurrent copy and kernel execution:          %s with %d copy "
            "engine(s)\n",
            (deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
    printf("  Run time limit on kernels:                     %s\n",
           deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
    printf("  Integrated GPU sharing Host Memory:            %s\n",
           deviceProp.integrated ? "Yes" : "No");
    printf("  Support host page-locked memory mapping:       %s\n",
           deviceProp.canMapHostMemory ? "Yes" : "No");
    printf("  Alignment requirement for Surfaces:            %s\n",
           deviceProp.surfaceAlignment ? "Yes" : "No");
    printf("  Device has ECC support:                        %s\n",
           deviceProp.ECCEnabled ? "Enabled" : "Disabled");

    printf("  CUDA Device Driver Mode (TCC or WDDM):         %s\n",
           deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)"
                                : "WDDM (Windows Display Driver Model)");

    printf("  Device supports Unified Addressing (UVA):      %s\n",
           deviceProp.unifiedAddressing ? "Yes" : "No");
    printf("  Device supports Managed Memory:                %s\n",
           deviceProp.managedMemory ? "Yes" : "No");
    printf("  Device supports Compute Preemption:            %s\n",
           deviceProp.computePreemptionSupported ? "Yes" : "No");
    printf("  Supports Cooperative Kernel Launch:            %s\n",
           deviceProp.cooperativeLaunch ? "Yes" : "No");
    printf("  Supports MultiDevice Co-op Kernel Launch:      %s\n",
           deviceProp.cooperativeMultiDeviceLaunch ? "Yes" : "No");
    printf("  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n",
           deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);

    const char *sComputeMode[] = {
            "Default (multiple host threads can use ::hipSetDevice() with device "
            "simultaneously)",
            "Exclusive (only one host thread in one process is able to use "
            "::hipSetDevice() with this device)",
            "Prohibited (no host thread can use ::hipSetDevice() with this "
            "device)",
            "Exclusive Process (many threads in one process is able to use "
            "::hipSetDevice() with this device)",
            "Unknown", nullptr };
    printf("  Compute Mode:\n");
    printf("     < %s >\n", sComputeMode[deviceProp.computeMode]);

    printf("\n");
    std::string sProfileString = "deviceQuery, CUDA Driver = CUDART";
    char cTemp[16];

    // driver version
    sProfileString += ", CUDA Driver Version = ";
    sprintf_s(cTemp, 10, "%d.%d", driverVersion / 1000,
              (driverVersion % 100) / 10);
    sProfileString += cTemp;

    // Runtime version
    sProfileString += ", CUDA Runtime Version = ";
    sprintf_s(cTemp, 10, "%d.%d", runtimeVersion / 1000,
              (runtimeVersion % 100) / 10);
    sProfileString += cTemp;

    printf("%s\n", sProfileString.c_str());

    printf("\n-------------------------- END OF DEVICE INFO --------------------------\n");
}
#pragma endregion

#pragma region Creating vectors
// Функция для создания вектора
double* createVector(int size) {
    auto* array = new double[size];       // Динамическая инициализация матрицы
    for (int i = 0; i < size; i++) {
        array[i] = i + 1.5;
    }
    return array;
}

// Процедура для удаления вектора
void deleteVector(const double* array) {
    delete[] array;
}
#pragma endregion

#pragma region Running CUDA
#pragma region Thread execution
__global__ void kernel(const double* A, double* B, const double* C, const double* D, double* Y, int size)
{
    // Условная координата в выходном векторе для текущего потока
    unsigned int idx_thread = blockIdx.x * blockDim.x + threadIdx.x;
    // Выполнение задачи
    for (int y = 0; y < size; y++)
    {
        Y[idx_thread * size + y] = (A[idx_thread * size + y] + C[idx_thread * size + y]) / (pow(B[idx_thread * size + y], m) - D[idx_thread * size + y]);
    }
}
#pragma endregion
#pragma region Init, run, free
void runCuda(const double* A, const double* B, const double* C, const double* D, double* Y, int size, int GRID_SIZE, int BLOCK_SIZE)
{
#pragma region Time measurement init
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#pragma endregion
#pragma region Pointers init
    double *d_A, *d_B, *d_C, *d_D, *d_Y;
    const int forSize = size / (GRID_SIZE * BLOCK_SIZE);
#pragma endregion
#pragma region Memory allocation
    gpuErrchk(hipMalloc<double>(&d_A, size));
    gpuErrchk(hipMalloc<double>(&d_B, size));
    gpuErrchk(hipMalloc<double>(&d_C, size));
    gpuErrchk(hipMalloc<double>(&d_D, size));
    gpuErrchk(hipMalloc<double>(&d_Y, size));
#pragma endregion
#pragma region Data copying
    gpuErrchk(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_C, C, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_D, D, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice));
#pragma endregion
#pragma region Setting sizes of grid&block
    const dim3 block(BLOCK_SIZE);                                   // Размер блока (одномерный)
    const dim3 grid(GRID_SIZE);                                     // Размер сетки из блоков (одномерная)
#pragma endregion
#pragma region Start CUDA
    hipEventRecord(start);                                         // Запуск таймера
    kernel <<< grid, block >>>(d_A, d_B, d_C, d_D, d_Y, forSize);   // Запуск процедуры
    gpuErrchk(hipPeekAtLastError());                               // Проверка на ошибки
    hipEventRecord(stop);                                          // Остановка таймера
#pragma endregion
#pragma region Result copying, free pointers
    hipMemcpy(Y, d_Y, size, hipMemcpyDeviceToHost);               // Копирование из памяти видеокарты в память процессора
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(d_Y);
#pragma endregion
#pragma region Time measurement. file output
    hipEventSynchronize(stop);                                     // Синхронизация таймера
    float time;                                                     // Время
    hipEventElapsedTime(&time, start, stop);                       // Расчёт времени
    ofstream gpuTime("../gpu.csv", std::ios::app);                  // Открытие файла
    cout << "\nProcessing time for GPU (ms): " << time << "\n";     // Вывод в поток
    // Вывод в файл
    gpuTime << size << "," << BLOCK_SIZE << "," << GRID_SIZE << "," << time << "\n";
    gpuTime.close();                                                // Закрытие файла
#pragma endregion
}
#pragma endregion
#pragma endregion

#pragma region Running CPU
void runCPU(const double* A, const double* B, const double* C, const double* D, double* Y, int size)
{
    for (int i = 0; i < size; i++) {
        Y[i] = (A[i] + C[i]) / (pow(B[i], m) - D[i]);

    }
}
#pragma endregion

int main() {
    // Вывод информации о GPU
    AboutDevice();
#pragma region Variables
    double* A, *B, *C, *D, *Y;                                      // Создание указателей векторов
    vector<int> sizes = {16384, 65536};                             // Список размеров векторов
    // Размеры блоков и их количество
    vector<pair<int,int>> blocks = {{1, 1},{1, 4}, {1, 32}, {1, 64}, {1, 256}, {1, 512},
                                    {2, 1}, {2, 4}, {2, 32}, {2, 64}, {2, 256}, {2, 512},
                                    {4, 1}, {4, 4}, {4, 32}, {4, 64}, {4, 256}, {4, 512},
                                    {8, 1}, {8, 4}, {8, 32}, {8, 64}, {8, 256}, {8, 512},
                                    {16, 1}, {16, 4}, {16, 32}, {16, 64}, {16, 256}, {16, 512}};
#pragma endregion
#pragma region File prep
    ofstream gpuTime("../gpu.csv", std::ios::app);
    ofstream cpuTime("../cpu.csv", std::ios::app);
    gpuTime << "Размер массивов, размер блока, количество блоков, время (мс)" << "\n";
    cpuTime << "Размер массивов, время (мс)" << "\n";
#pragma endregion
#pragma region Main programm
    cout << "\n-------------------------- PROCESSING STARTED --------------------------\n";
    for (const auto &size : sizes) {
#pragma region Vectors creation
        cout << "MATRIX SIZE: " << size << "\n";
        A = createVector(size);
        B = createVector(size);
        C = createVector(size);
        D = createVector(size);
        Y = new double[size];
        cout << "MATRICES CREATED\n";
#pragma endregion
#pragma region Run CUDA
        cout << "\n-------------------------- CUDA PART STARTED --------------------------\n";
        for (const auto &block : blocks) {
            runCuda(A, B, C, D, Y, size, block.first, block.second);
        }
        cout << "\n-------------------------- CUDA PART FINISHED --------------------------\n";
#pragma endregion
#pragma region Run CPU
        cout << "\n-------------------------- CPU PART STARTED --------------------------\n";
        double time = measureExecutionTime([&]() {runCPU(A, B, C, D, Y, size);});
        cout << "\nProcessing time for CPU (ms): " << time << "\n";
        cpuTime << size << "," << time << "\n";
        cout << "\n-------------------------- CPU PART FINISHED --------------------------\n";
#pragma endregion
#pragma region Memory free
        deleteVector(A);
        deleteVector(B);
        deleteVector(C);
        deleteVector(D);
        deleteVector(Y);
#pragma endregion
    }
    cout << "\n-------------------------- PROCESSING FINISHED --------------------------\n";
#pragma endregion
#pragma region File closing
    gpuTime.close();
    cpuTime.close();
#pragma endregion
    return 0;
}
